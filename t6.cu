
#include <hip/hip_runtime.h>
#include <cmath>
#include <string>
#include <cstdio>
#include <vector>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <hipblas.h>

using namespace std;

__global__ void sigmoid(float *arr, int size)
{
	int id = threadIdx.x;
	if(id < size && id >= 0) 
		arr[id] = 1 / (1 + exp(-arr[id]));
}

void check(float res){
	float real = round(0.0097 * 10000) / 10000;
	res = round(res * 10000) / 10000;
	if(real == res){
        cout << "correct"<< "\n";
    }else{
        cout << "result("<< res << ") != real(" << real << ")" << "\n";
    }
}

class Layer
{
private:
	hipblasHandle_t handle;
	float alpha, beta;
	float *weights, *biases;
	int in_size, out_size;

	void read_weights(string pathToWeights){
		float *h_arr = new float [in_size*out_size];
		float *host_array = new float [in_size*out_size];
		try{
			ifstream s(pathToWeights);
			for (int i = 0; i < in_size*out_size; i++){
                s >> h_arr[i];
            }
			s.close();
		}
		catch(exception const& e){
			cout << "error: " << e.what() << "\n";
		}
		for(int i=0;i<in_size;i++){
			for(int j=0;j<out_size;j++){
				host_array[i*out_size+j] = h_arr[j*in_size+i];
			}
		}
		hipMalloc(&weights, out_size * in_size * sizeof(float));
		hipMemcpy(weights, host_array, out_size * in_size * sizeof(float), hipMemcpyHostToDevice);
		delete[] h_arr;
		delete[] host_array;
	};

	void read_biases(string pathToWeights){
		float *h_arr = new float [out_size];
		try{
			ifstream s(pathToWeights);
			for (int i = 0; i < out_size; i++){
                s >> h_arr[i];
            }
			s.close();
		}
		catch(exception const& e){
			cout << "error: " << e.what() << "\n";
		}
		hipMalloc(&biases, out_size * sizeof(float));
		hipMemcpy(biases, h_arr, out_size*sizeof(float), hipMemcpyHostToDevice);
		delete[] h_arr;
	};

public:
	Layer(){
		in_size = 32 * 32;
		out_size = 1;
		alpha = 1.0;
		beta = 1.0;
	};

	Layer(string pathToWeights, string pathToBiases, int inSize, int outSize){
		alpha = 1.0;
		beta = 1.0;
		in_size = inSize;
		out_size = outSize;
		read_weights(pathToWeights);
		read_biases(pathToBiases);
	};

	float* Linear(float* input){
		hipblasCreate(&handle);
		hipblasSgemv(handle, HIPBLAS_OP_N, out_size, in_size, &alpha, weights,out_size, input, 1, &beta, biases, 1);
		hipblasDestroy(handle);
		sigmoid<<<1, out_size>>> (biases, out_size);
		return biases;
	};

	~Layer(){
		hipFree(weights);
		hipFree(biases);
	};
};

class Model
{
private:
	float *array;
	int in_size, out_size;



	void read(string path){
		float *inp_arr = new float [in_size];
		try{
			ifstream s(path);
			for (int i = 0; i < in_size; i++){
                s >> inp_arr[i];
            }
			s.close();
		}
		catch(exception const& e){
			cout << "error: " << e.what() << "\n";
		}
		hipMalloc(&array, in_size * sizeof(float));
		hipMemcpy(array, inp_arr, in_size*sizeof(float), hipMemcpyHostToDevice);
		delete[] inp_arr;
	};


	void print_res(float* arr){
		float* h_arr = new float[out_size];
		hipMemcpy(h_arr, arr, out_size*sizeof(float), hipMemcpyDeviceToHost);
		cout << "Result: " << "\n";
		for (int i = 0; i < out_size; i++){
			cout << h_arr[i] << "\n";
		}
		check(h_arr[0]);
		delete[] h_arr;
	};

public:
	Model(){
		in_size = 32 * 32;
		out_size = 1;
	};

	void forward(string path){
		read(path);
		Layer layer1("w1.bin", "b1.bin", 32 * 32, 16 * 16);
		array = layer1.Linear(array);
		Layer layer2("w2.bin", "b2.bin", 16 * 16, 4 * 4);
		array = layer2.Linear(array);
		Layer layer3("w3.bin", "b3.bin", 4 * 4, 1);
		array = layer3.Linear(array);
		print_res(array);
	}

	~Model(){
		hipFree(array);
	};
};

int main()
{
	Model model;
	model.forward("inp.bin");
	return 0;
}